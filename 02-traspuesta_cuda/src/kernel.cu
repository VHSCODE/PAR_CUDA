#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>

#include "../include/kernel.cuh"

#define TAMAÑO_BLOQUE 16

#include "../include/toggles.h"
namespace CUDA {


    template<typename T>
    __global__ void transpose(const T *A, T * DST,Dimensiones dimA)
    {
        long fila = blockIdx.y * blockDim.y + threadIdx.y;
        long columna = blockIdx.x * blockDim.x + threadIdx.x;

        if(columna < dimA.width && fila < dimA.height)
        {
            DST[columna * dimA.height + fila] = A[columna + fila * dimA.width];
        }

    }
    void transponer_cuda(const float *A , float *DST, Dimensiones dimA, float * tiempoEjecucion)
    {
        //Matrices de la GPU
        float *A_GPU;
        float *DST_GPU;

        //Asignamos memoria a las matrices
        if(hipMalloc(&A_GPU,sizeof (float) * dimA.width * dimA.height) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz A_GPU\n");
            return;
        }
        if(hipMalloc(&DST_GPU,sizeof (float) * dimA.width * dimA.height) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz A_GPU\n");
            return;
        }

        //Y copiamos los valores de las matrices del host
        if(hipMemcpy(A_GPU,A,sizeof (float) * dimA.width * dimA.height,hipMemcpyHostToDevice) != hipSuccess){
            printf("[ERROR] No se ha podido copiar el objeto A a A_GPU\n");
            return;
        }

        dim3 DimensionesBloque(TAMAÑO_BLOQUE,TAMAÑO_BLOQUE);
        dim3 DimensionesGrid( (dimA.width + DimensionesBloque.x - 1) / DimensionesBloque.x, (dimA.height + DimensionesBloque.x - 1) / DimensionesBloque.x);


#ifdef DEBUG
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
#endif
        transpose<<<DimensionesGrid,DimensionesBloque>>>(A_GPU,DST_GPU,dimA);

#ifdef DEBUG
        hipEventRecord(stop);
#endif

        if(hipMemcpy(DST,DST_GPU,sizeof(float) * dimA.width * dimA.height,hipMemcpyDeviceToHost) != hipSuccess){
            printf("[ERROR] No se ha podido copiar la matrix DST al host\n");
            return;
        }
#ifdef DEBUG
        hipEventSynchronize(stop);

        hipEventElapsedTime(tiempoEjecucion,start,stop);
#endif

        //Esperamos que se terminen los procesos en la gpu
        hipDeviceSynchronize();



        //Liberamos la memoria
        hipFree(A_GPU);
        hipFree(DST_GPU);
        
    }

}