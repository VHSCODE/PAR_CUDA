#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstdio>

#include "../include/kernel.cuh"

#include "../include/toggles.h"
#define TAMAÑO_BLOQUE 16
#include "../include/MatrixUtils.h"
namespace CUDA {


    template<typename T>
    __global__ void multiplicar_cuda(const T *A, const T *B, T * DST,Dimensiones dimA, Dimensiones dimB, Dimensiones dimDST)
    {

        //Suma parcial
        T suma = 0;

        long fila = blockIdx.y * TAMAÑO_BLOQUE + threadIdx.y;
        long columna = blockIdx.x * TAMAÑO_BLOQUE + threadIdx.x;

        //Usaremos estas caches para guardar los valores, lo usaran todos los threads
        __shared__ T A_cache[TAMAÑO_BLOQUE][TAMAÑO_BLOQUE];
        __shared__ T B_cache[TAMAÑO_BLOQUE][TAMAÑO_BLOQUE];


        //En este bucle vamos iterando por las submatrices de A y B y guardando los valores necesarios
        for(int i = 0; i < (TAMAÑO_BLOQUE + dimA.width -1)/TAMAÑO_BLOQUE; i++){

            if(i * TAMAÑO_BLOQUE + threadIdx.x < dimA.width && fila < dimA.height)
                A_cache[threadIdx.y][threadIdx.x] = A[fila * dimA.width + i * TAMAÑO_BLOQUE + threadIdx.x];
            else
                A_cache[threadIdx.y][threadIdx.x] = 0.0;

            if (i*TAMAÑO_BLOQUE + threadIdx.y < dimB.height && columna < dimB.width)
                B_cache[threadIdx.y][threadIdx.x] = B[(i * TAMAÑO_BLOQUE + threadIdx.y) * dimB.width + columna];
            else
                B_cache[threadIdx.y][threadIdx.x] = 0.0;

            __syncthreads(); //Sincronizamos para que todos los thread terminen sus calculos



            //Multiplicamos las matrices parciales
            for (int n = 0; n < TAMAÑO_BLOQUE; ++n)
                suma += A_cache[threadIdx.y][n] * B_cache[n][threadIdx.x];

            __syncthreads(); //Volvemos a sincronizar
        }


        //Finalmente asignamos el valor calculado
        if(fila < dimDST.height && columna < dimDST.width)
        DST[((blockIdx.y * blockDim.y + threadIdx.y)*dimDST.width) + (blockIdx.x * blockDim.x)+ threadIdx.x] = suma;



    }

    template<typename T>
    __global__ void sumar_cuda(const T *A, const T *B, T *DST,long N)
    {
        long tid = (blockDim.x * blockIdx.x) + threadIdx.x;
        if (tid < N) {
            DST[tid] = A[tid] + B[tid];
        }
    }


    void matmuladd_calcular(const float *A, const float *B, const float *C, float *R, Dimensiones dimA, Dimensiones dimB, Dimensiones dimC,float *tiempoEjecucion)
    {

        if (dimA.width != dimB.height) {
            printf("[ERROR] Las dimensiones introducidas no son validas para hacer el calculo\n");

            return;
        }

        //Matrices de la GPU
        float *A_GPU, *B_GPU, *C_GPU, *MUL_GPU, * R_GPU;


        //Asignamos memoria a las matrices
        if(hipMalloc(&A_GPU,sizeof (float) * dimA.width * dimA.height) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz A_GPU\n");
            return;
        }
        if(hipMalloc(&B_GPU,sizeof (float) * dimB.width * dimB.height) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz B_GPU\n");
            return;
        }
        if(hipMalloc(&C_GPU,sizeof (float) * dimC.width * dimC.height) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz C_GPU\n");
            return;
        }

        Dimensiones dimR{dimA.height, dimB.width};

        if(hipMalloc(&MUL_GPU, sizeof (float) * dimR.height * dimR.width) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz MUL_GPU\n");
            return;
        }
        if(hipMalloc(&R_GPU,sizeof (float) * dimR.height * dimR.width) != hipSuccess){
            printf("[ERROR] No se ha podido reservar memoria la matriz C_GPU\n");
            return;
        }



        //Y copiamos los valores de las matrices del host
        if(hipMemcpy(A_GPU,A,sizeof (float) * dimA.width * dimA.height,hipMemcpyHostToDevice) != hipSuccess){
            printf("[ERROR] No se ha podido copiar el objeto A a A_GPU\n");
            return;
        }


        if(hipMemcpy(B_GPU,B,sizeof (float) * dimB.width * dimB.height,hipMemcpyHostToDevice) != hipSuccess){
            printf("[ERROR] No se ha podido copiar el objeto A a B_GPU\n");
            return;
        }
        if(hipMemcpy(C_GPU,C,sizeof (float) * dimC.width * dimC.height,hipMemcpyHostToDevice) != hipSuccess){
            printf("[ERROR] No se ha podido copiar el objeto A a C_GPU\n");
            return;
        }

        //Creamos las dimensiones del grid y de los bloques
        dim3 dimensionesBloque(TAMAÑO_BLOQUE, TAMAÑO_BLOQUE);

        dim3 dimensionesGrid((dimB.height + TAMAÑO_BLOQUE - 1)/TAMAÑO_BLOQUE, (dimA.width+ TAMAÑO_BLOQUE-1)/TAMAÑO_BLOQUE);

        long N = dimC.width * dimC.height;

#ifdef DEBUG
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
#endif
        // Y comenzamos a llamar a los kernels
        multiplicar_cuda<<<dimensionesGrid,dimensionesBloque>>>(A_GPU, B_GPU, MUL_GPU,
                                                                dimA, dimB, dimR);


        dim3 dimGrid((N + TAMAÑO_BLOQUE -1 ) / TAMAÑO_BLOQUE);
        sumar_cuda<<<dimGrid, dimensionesBloque>>>(C_GPU,MUL_GPU, R_GPU,N );


#ifdef DEBUG
        hipEventRecord(stop);
        hipEventSynchronize(stop);
#endif
        //Esperamos que se terminen los procesos en la gpu
        hipDeviceSynchronize();
#ifdef DEBUG


        hipEventElapsedTime(tiempoEjecucion,start,stop);
#endif


        auto err = hipMemcpy(R, R_GPU, sizeof(float) * dimR.height * dimR.width, hipMemcpyKind::hipMemcpyDeviceToHost);
        //Escribimos el resultado en la matriz host
        if ( err != hipSuccess)
        {

            printf("[ERROR] Error al copiar la matriz R de la GPU al host : %s \n", hipGetErrorName(err));
            return;
        }




        //Liberamos la memoria
        hipFree(A_GPU);
        hipFree(B_GPU);
        hipFree(C_GPU);
        hipFree(MUL_GPU);
        hipFree(R_GPU);
    }

}